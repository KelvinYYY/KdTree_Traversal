#include "hip/hip_runtime.h"



#include "CUDA_KDtree.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <cstdio>
#include ""
#include <iostream>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "math.h"

#define CUDA_STACK 10000 // fixed size stack elements for each thread, increase as required. Used in SearchAtNodeRange.
#define STACK_SIZE 50000
void CheckCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__device__ float Distance(const Point &a, const Point &b)
{
    float dist = 0;

    for(int i=0; i < KDTREE_DIM; i++) {
        float d = a.coords[i] - b.coords[i];
        dist += d*d;
    }

    return dist;
}

__device__ void SearchAtNode(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int cur, const Point &query, int *ret_index, float *ret_dist, int *ret_node, int mode, int* stack, int query_size)
{
    // Finds the first potential candidate
	//query is the current point
	//nodes are the treenodes
	//query is the point
    int* s1 = &stack[0];	
    int best_idx = 0;
    float best_dist = FLT_MAX;
	if (mode == 0){
		s1[0] = 0;
		int top1 = 0;
		while (top1 != -1){
			cur = s1[top1];
			top1--;
			int split_axis = nodes[cur].level % KDTREE_DIM;

			if (nodes[cur].left == -1){//if it is leaf, update best distance

				for (int i = 0; i < nodes[cur].num_indexes; i++) {
					int idx = indexes[nodes[cur].indexes + i];
					float dist = Distance(query, pts[idx]);
					if (dist < best_dist) {
						best_dist = dist;
						best_idx = idx;
					}
				}

				continue;
			}
			else if (query.coords[split_axis] < nodes[cur].split_value){  //closer to left

				if ((query.coords[split_axis] - nodes[cur].split_value)*(query.coords[split_axis] - nodes[cur].split_value) <= best_dist){
					if (nodes[cur].right != -1){
						s1[++top1] = nodes[cur].right;
					}
				}
				else{
					//trancated
				}
				s1[++top1] = nodes[cur].left;

			}
			else{

				if ((query.coords[split_axis] - nodes[cur].split_value)*(query.coords[split_axis] - nodes[cur].split_value) <= best_dist){
					s1[++top1] = nodes[cur].left;
				}
				if (nodes[cur].right != -1){
					s1[++top1] = nodes[cur].right;
				}

			}
		}
	}
	if (mode != 0){	
		unsigned int s1[CUDA_STACK];
		unsigned int s2[CUDA_STACK];
		int top1 = 0;
		int top2 = 0;
		int cur = 0;
		s1[0] = 0;
		s2[0] = 1;
		unsigned int warp_mask;
		int split_axis,a,b,i;
		while (top1 > -1 && top2>-1){
			cur = s1[top1];
			top1--;
			warp_mask = s2[top2];
			top2--;
			
			split_axis = nodes[cur].level % KDTREE_DIM;
		

			if (warp_mask== 1){
				//if thread in warp is active
				if (nodes[cur].level - 1 % KDTREE_DIM >= 0 && nodes[cur].parent >= 0){

					a = query.coords[nodes[cur].level - 1 % KDTREE_DIM] - nodes[nodes[cur].parent].split_value;
					b = pts[indexes[nodes[cur].indexes]].coords[nodes[cur].level - 1 % KDTREE_DIM] - nodes[nodes[cur].parent].split_value;
					if(a*b < 0 && a*a>best_dist*best_dist){
						warp_mask = 0;
					}

				}
				if (nodes[cur].left == -1){
					//if it is leaf, update correlation
					for (i = 0; i < nodes[cur].num_indexes; i++) {
						int id = indexes[nodes[cur].indexes + i];
						float dist = Distance(query, pts[id]);
						if (dist < best_dist) {
							
							best_dist = dist;
							best_idx = id;
						}
					}
					warp_mask = 0;
					
				 }

			}
			//combine mask from all threads in warp

			if (__any(warp_mask != 0)){
				if (nodes[cur].right != -1){
						s1[++top1] = nodes[cur].right;
						if (query.coords[split_axis] < nodes[cur].split_value && (query.coords[split_axis] - nodes[cur].split_value)*(query.coords[split_axis] - nodes[cur].split_value) > best_dist){
							//closer to left and query point distance to split axis already big enough, trancate right side.
							s2[++top2] = 0;
						}
						else{
							s2[++top2] = warp_mask;
						}
				}
				s1[++top1] = nodes[cur].left;
				if (query.coords[split_axis] > nodes[cur].split_value && (query.coords[split_axis] - nodes[cur].split_value)*(query.coords[split_axis] - nodes[cur].split_value) > best_dist){
					//if closer to right
					s2[++top2] = 0;
				}
				else{
					s2[++top2] = warp_mask;
				}
			}
		}
	}//mode!=0 end bracket
	
	*ret_index = best_idx;
	*ret_dist = best_dist;
}


__device__ void Search(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, int *ret_index, float *ret_dist, int mode, int* stack)
{
    int best_node= 0; 
    int best_idx = 0;
    float best_dist = FLT_MAX;
    SearchAtNode(nodes, indexes, pts, 0 /* root */, query, &best_idx, &best_dist, &best_node, mode, stack);
	if (mode == 1){
		*ret_index = best_idx;
		*ret_dist = best_dist;
		return;
	}
	if (mode == 0){
		*ret_index = best_idx;
		*ret_dist = best_dist;
		return;
	}

}

__global__ void SearchBatch(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int num_pts, Point *queries, int num_queries, int *ret_index, float *ret_dist, int mode, int* stack)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
	

    if(idx >= num_queries)
        return;

    Search(nodes, indexes, pts, queries[idx], &ret_index[idx], &ret_dist[idx], mode, &stack[idx], num_queries);
}
	


__device__ CUDA_KDTree::~CUDA_KDTree()
{
    hipFree(m_gpu_nodes);
    hipFree(m_gpu_indexes);
    hipFree(m_gpu_points);
}

void CUDA_KDTree::CreateKDTree(KDNode *root, int num_nodes, const vector <Point> &data)
{
    // Create the nodes again on the CPU, laid out nicely for the GPU transfer
    // Not exactly memory efficient, since we're creating the entire tree again
    m_num_points = data.size();

    hipMalloc((void**)&m_gpu_nodes, sizeof(CUDA_KDNode)*num_nodes);
    hipMalloc((void**)&m_gpu_indexes, sizeof(int)*m_num_points);
    hipMalloc((void**)&m_gpu_points, sizeof(Point)*m_num_points);
    CheckCUDAError("CreateKDTree");

    vector <CUDA_KDNode> cpu_nodes(num_nodes);
    vector <int> indexes(m_num_points);
    vector <KDNode*> to_visit;

    int cur_pos = 0;

    to_visit.push_back(root);

    while(to_visit.size()) {
        vector <KDNode*> next_search;

        while(to_visit.size()) {
            KDNode *cur = to_visit.back();
            to_visit.pop_back();

            int id = cur->id;

            cpu_nodes[id].level = cur->level;
            cpu_nodes[id].parent = cur->_parent;
            cpu_nodes[id].left = cur->_left;
            cpu_nodes[id].right = cur->_right;
            cpu_nodes[id].split_value = cur->split_value;
            cpu_nodes[id].num_indexes = cur->indexes.size();

            if(cur->indexes.size()) {
                for(unsigned int i=0; i < cur->indexes.size(); i++)
                    indexes[cur_pos+i] = cur->indexes[i];

                cpu_nodes[id].indexes = cur_pos;
                cur_pos += cur->indexes.size();
            }


            else {
                cpu_nodes[id].indexes = -1;
            }

            if(cur->left)
                next_search.push_back(cur->left);

            if(cur->right)
                next_search.push_back(cur->right);
        }

        to_visit = next_search;
    }

    hipMemcpy(m_gpu_nodes, &cpu_nodes[0], sizeof(CUDA_KDNode)*cpu_nodes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_indexes, &indexes[0], sizeof(int)*indexes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_points, &data[0], sizeof(Point)*data.size(), hipMemcpyHostToDevice);
 
    CheckCUDAError("CreateKDTree");
}

void CUDA_KDTree::Search(const vector <Point> &queries, vector <int> &indexes, vector <float> &dists, int mode)
{
	int threads;
		 threads = 512;
    int blocks = queries.size()/threads + ((queries.size() % threads)?1:0);
    
    Point *gpu_queries;
    int *gpu_ret_indexes;
    float *gpu_ret_dist;
    int* gpu_stack;
    int* gpu_mask;
    indexes.resize(queries.size());
    dists.resize(queries.size());

    hipMalloc((void**)&gpu_queries, sizeof(Point)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_indexes, sizeof(int)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_dist, sizeof(float)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_stack, sizeof(int)*queries.size()*STACK_SIZE*2);
    CheckCUDAError("Search");

    hipMemcpy(gpu_queries, &queries[0], sizeof(float)*queries.size()*KDTREE_DIM, hipMemcpyHostToDevice);

    CheckCUDAError("Search");

    printf("CUDA blocks/threads: %d %d\n", blocks, threads);
    SearchBatch<<<blocks, threads >>>(m_gpu_nodes, m_gpu_indexes, m_gpu_points, m_num_points, gpu_queries, queries.size(), gpu_ret_indexes, gpu_ret_dist, mode,gpu_stack);
	
    hipDeviceSynchronize();
    CheckCUDAError("Search");

    hipMemcpy(&indexes[0], gpu_ret_indexes, sizeof(int)*queries.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&dists[0], gpu_ret_dist, sizeof(float)*queries.size(), hipMemcpyDeviceToHost);

    hipFree(gpu_queries);
    hipFree(gpu_ret_indexes);
    hipFree(gpu_ret_dist);
   
    hipFree(gpu_stack);
    hipFree(gpu_mask);
}
